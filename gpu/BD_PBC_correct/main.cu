#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h> 
#include <time.h>
#include <string.h>
#include <math.h>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


#define G 1 // gravitational constant
#define delta_t 0.1 // timestep of integration
int total_time=5000; // time of integration
double rangex=0,rangey=0,rangez=0; // ranges in x and y and z direction
#define row 6 // no of rows
#define col 6 // no of columns
int boxcount=0; // count the boxes
int boxes;
double width=10; // width of each box
double height=10; // height of each box 
double depth=60; // depth of each box
double widthc=10; // width of each box copy
double heightc=10; // height of each box copy
double depthc=60; // depth of each box copy
double rad=0.5; // modify boxes according to the radius of the particle
double startx=0;
double starty=0;
double startz=0;
int npart; // no of particles
int sumpart=0; // sum of the no of particles
#define N 36 // launch cuda cores = no of boxes in the cell
double arr_startx[N],arr_starty[N],arr_startz[N];
double arr_width[N],arr_height[N],arr_depth[N]; 
int arr_startrow[N],arr_lastrow[N];

#include "vectdefs.h"
int x[NDIM]; // boxvector in x-direction
int y[NDIM]; // boxvector in y-direction
int z[NDIM]; // boxvector in z-direction
#include "maths.c"
#include "force.c"
#include "verlet.c"

int main()
{
 boxes=row*col; // total no of boxes
 int i=0,j=0,temporary,seed=6,k_2=0,k_3=0,k_4=0,count=0,colcount=0;
 int t=0;
 //int startrow=0,lastrow=0;
 double xr; // xr will store the random number
 FILE *fp;
 fp=fopen("output.xyz","w+"); // output coordinate file
 int rearr=0,count_core=0;

/*************************************** Boundary boxes ***************************************************************************************/
 
 double x_interval[4];
 double y_interval[4];
 x_interval[0]=startx;
 y_interval[0]=starty;
 x_interval[1]=width+0.5; // Make transitions continuous
 y_interval[1]=height+0.5;
 double calx;
 calx=col*width+(col-1);
 double caly;
 caly=row*height+(row-1);
 x_interval[2]=calx-width-0.5;x_interval[3]=calx;
 y_interval[2]=caly-height-0.5;y_interval[3]=caly;

 printf("x_interval\n");
 for(i=0;i<4;i++)
 printf("%f ",x_interval[i]);
 printf("\n");
 printf("y_interval\n");
 for(i=0;i<4;i++)
 printf("%f ",y_interval[i]);
 printf("\n");

/*************************** Dynamically allocate particle matrix which stores no of particles of boxes ***************************************/
 int *particle = (int *)malloc(boxes*sizeof(int));
 for(i=0;i<boxes;i++)
 {
  srand(seed);
  temporary = (rand() % 10);
  if(temporary==0) // npart should not be 0 and try to keep the min no of particles to be 2
  temporary+=2;
  else if(temporary==1)
  temporary++;
  particle[i]=temporary; 
  seed++;
 }
/****************************************** Print particle matrix  ***************************************************************************/
 
 printf("Printing particle matrix --------------------------\n");
 for(i=0;i<boxes;i++)
 printf("%d ",particle[i]);
 printf("\n");

/****************************************** Allocate arr_startrow ************************************************************/
 arr_startrow[0]=0;
 for(i=1;i<N;i++)
 {
  arr_startrow[i]=arr_startrow[i-1]+particle[i-1];
 }

 printf("Printing arr_startrow--------\n");
 for(i=0;i<N;i++)
 printf("%d ",arr_startrow[i]);
 printf("\n");

/****************************************** Sum particle matrix  *****************************************************************************/
 for(i=0;i<boxes;i++)
 sumpart+=particle[i];
 //printf("%d\n",sumpart);
 npart=sumpart;
/****************************************** Allocate arr_lastrow ***************************************************************************/
 arr_lastrow[N-1]=npart-1;
 for(i=N-2;i>=0;i--)
 arr_lastrow[i]=arr_lastrow[i+1]-particle[i+1];
 for(i=0;i<N;i++)
 arr_lastrow[i]=arr_lastrow[i]+1;
 
 printf("Printing arr_lastrow--------\n");
 for(i=0;i<N;i++)
 printf("%d ",arr_lastrow[i]);
 printf("\n");

/****************************************** Dynamically allocate entire box position matrix **************************************************/
 double **position = (double **)malloc(sumpart*sizeof(double *));
 for(i=0;i<sumpart;i++)
 position[i]= (double *)malloc(5*sizeof(double));
 for(i=0;i<sumpart;i++)
 {
  for(j=0;j<5;j++)
  position[i][j]=0;
 }

/****************************************** Dynamically allocate entire force  matrix **************************************************/

 double **force = (double **)malloc(sumpart*sizeof(double *)); // dynamically allocate the force matrix
 for(i=0;i<sumpart;i++)
 force[i]= (double *)malloc(5*sizeof(double));
 for(i=0;i<sumpart;i++) // initialize the force matrix to 0
 {
  for(j=0;j<5;j++)
  {
   force[i][j]=0;
  }
 }

/****************************************** Dynamically allocate entire velocity matrix **************************************************/

 double **velocity = (double **)malloc(sumpart*sizeof(double *)); // dynamically allocate the velocity matrix
 for(i=0;i<sumpart;i++)
 velocity[i]= (double *)malloc(5*sizeof(double));
 for(i=0;i<sumpart;i++) // initialize the velocity matrix to 0
 {
  for(j=0;j<5;j++)
  {
   velocity[i][j]=0;
  }
 }

/****************************************** Dynamically allocate entire halfvelocity matrix **************************************************/

 double **halfvelocity = (double **)malloc(sumpart*sizeof(double *)); // dynamically allocate the half-velocity matrix
 for(i=0;i<sumpart;i++)
 halfvelocity[i]= (double *)malloc(5*sizeof(double));
 for(i=0;i<sumpart;i++) // initialize the half-velocity matrix to 0
 {
  for(j=0;j<5;j++)
  {
   halfvelocity[i][j]=0;
  }
 }

/****************************************** Fill the 1st column of the position matrix ****************************************************/
 boxcount=0;j=0;
 while(boxcount<boxes)
 {
  for(i=0;i<particle[boxcount];i++)
  {
   position[j][0]=boxcount;
   force[j][0]=boxcount;
   velocity[j][0]=boxcount;
   halfvelocity[j][0]=boxcount;
   j++;
  }
  boxcount++;
 }
/****************************************** Fill the 2nd column of the position matrix ****************************************************/
 j=0;
 for(i=0;i<sumpart;i++)
 {
  position[j][1]=i;
  force[j][1]=i;
  velocity[j][1]=i;
  halfvelocity[j][1]=i;
  j++;
 }
/************************************************* Test print the position matrix ************************************************************/ 
 /*for(i=0;i<sumpart;i++)
 {
  for(j=0;j<5;j++)
  printf("%f ",position[i][j]);
  printf("\n");
 } 
 printf("\n\n");*/  
/****************************************** Generate random coordinates for all the boxes ****************************************************/

 i=0;boxcount=0;k_2=0;k_3=0;k_4=0;
 while(i<row)
 {
  colcount=0;
  while(colcount<col)
  {
   x[0]=startx;x[1]=width;x[2]=0;
   y[0]=starty;y[1]=height;y[2]=0;
   z[0]=startz;z[1]=depth;z[2]=0;
   rangex=abs(x[1]-x[0]);
   rangey=abs(y[1]-y[0]);
   rangez=abs(z[1]-z[0]);
   for(j=0;j<particle[boxcount];j++) // generate x-coordinates
   {
    xr = (float)rand()/(float)(RAND_MAX/rangex)+x[0];
    position[k_2][2]=xr;k_2++;
   }
   for(j=0;j<particle[boxcount];j++) // generate y-coordinates
   {
    xr = (float)rand()/(float)(RAND_MAX/rangey)+y[0];
    position[k_3][3]=xr;k_3++;
   }
   for(j=0;j<particle[boxcount];j++) // generate z-coordinates
   {
    xr = (float)rand()/(float)(RAND_MAX/rangez)+z[0];
    position[k_4][4]=xr;k_4++;
   }
   boxcount++;
   colcount++;
   startx=width+1;width=startx+widthc;
  }
  i++;startx=0;width=widthc;starty=height+1;height=starty+heightc;  
 }
/************************************************* Test print the position matrix ************************************************************/ 
 
 for(i=0;i<sumpart;i++)
 {
  for(j=0;j<5;j++)
  printf("%f ",position[i][j]);
  printf("\n");
 }
 
/************************************************* Write the file ************************************************************/
 fprintf(fp,"%d\n",sumpart); 
 for(i=0;i<sumpart;i++)
 {
  fprintf(fp,"Atom%d ",i);
  fprintf(fp,"%f %f %f\n",position[i][2],position[i][3],position[i][4]);
 }  

/*************************** Dynamically allocate organelle_pos matrix which stores coordinates of organelles ***********************************/
 /*double **organelle_pos = (double **)malloc(npart*sizeof(double *));
 for(i=0;i<npart;i++)
 organelle_pos[i]= (double *)malloc(5*sizeof(double));

 for(i=0;i<npart;i++)
 {
  for(j=0;j<5;j++)
  organelle_pos[i][j]=0;
 }*/

/*************************Dynamically allocate mass matrix which stores masses of particles ***************************************/
 double *mass = (double *)malloc(sumpart*sizeof(double));
 for(i=0;i<sumpart;i++)
 mass[i]=1; // all the particles are of mass 1

 //mass[1]=999; // organelle
 /*mass[7]=999;
 mass[12]=999;
 mass[15]=999;*/

/*************************** Dynamically allocate radius matrix which stores radii of particles **************************************/
 double *radius = (double *)malloc(sumpart*sizeof(double));
 for(i=0;i<sumpart;i++)
 radius[i]=0.5; // all the particles are of radius 0.5

 //radius[1]=1.0; // organelle
 /*radius[7]=1.0;
 radius[12]=1.0;
 radius[15]=1.0;*/

/*************************** Dynamically allocate organelle matrix which stores the organelle info **************************************/
 /*double *organelle = (double *)malloc(npart*sizeof(double));
 for(i=0;i<npart;i++)
 organelle[i]=0; // initialise the organelle array to 0 i.e., no organelles are present

 organelle[1]=1;*/ // this particle is an organelle
 /*organelle[7]=1;
 organelle[12]=1;
 organelle[15]=1;*/
 //organelle[5]=1;
 //organelle[6]=1;
 //organelle[11]=1;

/******************************** store coordinates of organelles into organelle_pos matrix ***********************************/
 /*for(i=0;i<npart;i++)
 {
  if(organelle[i]==1)
  {
   organelle_pos[i][0]=position[i][0];
   organelle_pos[i][1]=position[i][1];
   organelle_pos[i][2]=position[i][2];
   organelle_pos[i][3]=position[i][3];
   organelle_pos[i][4]=position[i][4];
  }
 }*/

/*************************** Generate the force matrix *******************************************************************************/
 
 force=vanderwalforce(force,position,mass,npart); // to calculate vanderwal force
 printf("\n\n FORCE MATRIX ----------------\n");
 for(i=0;i<sumpart;i++)
 {
  for(j=0;j<5;j++)
  printf("%f ",force[i][j]);
  printf("\n");
 }

/************************* Allocate arr_startx,arr_starty,arr_startz,arr_width,arr_height,arr_depth *****************************************************/
 i=0;boxcount=0;startx=0;width=widthc;starty=0;height=heightc;startz=0;depth=depthc;
 while(i<row)
 {
  colcount=0;
  while(colcount<col)
  {
   arr_startx[boxcount]=startx;
   arr_starty[boxcount]=starty;
   arr_startz[boxcount]=startz;
   arr_width[boxcount]=width;
   arr_height[boxcount]=height;
   arr_depth[boxcount]=depth;
   boxcount++;
   colcount++;
   startx=width+1;width=startx+widthc;
  }
  i++;startx=0;width=widthc;starty=height+1;height=starty+heightc; 
 }

 for(i=0;i<N;i++) // making transitions between boxes continuous
 {
  if((i%col)==0)
  continue;
  else
  arr_startx[i]-=0.5;
 }
  
 for(i=0;i<N;i++)
 {
  if((i%col)==(col-1))
  continue;
  else
  arr_width[i]+=0.5;
 }

  
 for(i=col;i<N;i++)
 arr_starty[i]-=0.5;

 for(i=0;i<N-col;i++)
 arr_height[i]+=0.5;
  

 printf("Printing arr_startx-------------------------\n");
 for(i=0;i<N;i++)
 printf("%f ",arr_startx[i]);
 printf("\n");
 printf("Printing arr_starty-------------------------\n");
 for(i=0;i<N;i++)
 printf("%f ",arr_starty[i]);
 printf("\n");
 printf("Printing arr_startz-------------------------\n");
 for(i=0;i<N;i++)
 printf("%f ",arr_startz[i]);
 printf("\n");
 printf("Printing arr_width-------------------------\n");
 for(i=0;i<N;i++)
 printf("%f ",arr_width[i]);
 printf("\n");
 printf("Printing arr_height-------------------------\n");
 for(i=0;i<N;i++)
 printf("%f ",arr_height[i]);
 printf("\n");
 printf("Printing arr_depth-------------------------\n");
 for(i=0;i<N;i++)
 printf("%f ",arr_depth[i]);
 printf("\n");

 











/************************* Convert main 2D arrays to 1D arrays to ease cuda processing ******************************************************************/
 
 double *scale_halfvelocity;// 1D scaled version of halfvelocity        host copies
 double *scale_velocity;// 1D scaled version of velocity
 double *scale_force;// 1D scaled version of force
 double *scale_position;// 1D scaled version of position
 double *dup_scale_position,*dup_scale_halfvelocity,*dup_scale_velocity,*dup_scale_force;

 scale_halfvelocity=(double *)malloc(5*sumpart*sizeof(double));
 scale_velocity=(double *)malloc(5*sumpart*sizeof(double));
 scale_force=(double *)malloc(5*sumpart*sizeof(double));
 scale_position=(double *)malloc(5*sumpart*sizeof(double));
 dup_scale_position=(double *)malloc(5*sumpart*sizeof(double));
 dup_scale_halfvelocity=(double *)malloc(5*sumpart*sizeof(double));
 dup_scale_velocity=(double *)malloc(5*sumpart*sizeof(double));
 dup_scale_force=(double *)malloc(5*sumpart*sizeof(double));



 
 count=0;
 for(i=0;i<sumpart;i++) // Allocating halfvelocity matrix to scale_halfvelocity array
 {
  for(j=0;j<5;j++)
  {
   scale_halfvelocity[count]=halfvelocity[i][j];
   count++;
  }
 }


 
 count=0;
 for(i=0;i<sumpart;i++) // Allocating velocity matrix to scale_velocity array
 {
  for(j=0;j<5;j++)
  {
   scale_velocity[count]=velocity[i][j];
   count++;
  }
 }

 count=0;
 for(i=0;i<sumpart;i++) // Allocating force matrix to scale_force array
 {
  for(j=0;j<5;j++)
  {
   scale_force[count]=force[i][j];
   count++;
  }
 }

 count=0;
 for(i=0;i<sumpart;i++) // Allocating position matrix to scale_position array
 {
  for(j=0;j<5;j++)
  {
   scale_position[count]=position[i][j];
   count++;
  }
 }

 count=0;
 for(i=0;i<sumpart;i++) // Initialising dup_scale_position 
 {
  for(j=0;j<5;j++)
  {
   dup_scale_position[count]=0;
   count++;
  }
 }

 count=0;
 for(i=0;i<sumpart;i++) // Initialising dup_scale_halfvelocity 
 {
  for(j=0;j<5;j++)
  {
   dup_scale_halfvelocity[count]=0;
   count++;
  }
 }

 count=0;
 for(i=0;i<sumpart;i++) // Initialising dup_scale_velocity 
 {
  for(j=0;j<5;j++)
  {
   dup_scale_velocity[count]=0;
   count++;
  }
 }

 count=0;
 for(i=0;i<sumpart;i++) // Initialising dup_scale_force 
 {
  for(j=0;j<5;j++)
  {
   dup_scale_force[count]=0;
   count++;
  }
 }

 free(halfvelocity);free(velocity);free(force);free(position);


/************************* Device initialisations and allocating space **********************************************************************************/
 //hipDeviceReset();
 double *dev_halfvelocity,*dev_velocity,*dev_force,*dev_position,*dev_mass,*dev_radius;// device copies
 int *dev_arr_startrow,*dev_arr_lastrow;
 double *dev_arr_startx,*dev_arr_starty,*dev_arr_startz,*dev_arr_width,*dev_arr_height,*dev_arr_depth;
 double *dev_x_interval,*dev_y_interval;

 hipMalloc((void **) &dev_halfvelocity,5*sumpart*sizeof(double));
 hipMalloc((void **) &dev_velocity,5*sumpart*sizeof(double));
 hipMalloc((void **) &dev_force,5*sumpart*sizeof(double));
 hipMalloc((void **) &dev_position,5*sumpart*sizeof(double));
 hipMalloc((void **) &dev_mass,1*sumpart*sizeof(double));
 hipMalloc((void **) &dev_radius,1*sumpart*sizeof(double));
 hipMalloc((void **) &dev_arr_startrow,N*sizeof(int));
 hipMalloc((void **) &dev_arr_lastrow,N*sizeof(int));
 hipMalloc((void **) &dev_arr_startx,N*sizeof(double));
 hipMalloc((void **) &dev_arr_starty,N*sizeof(double));
 hipMalloc((void **) &dev_arr_startz,N*sizeof(double));
 hipMalloc((void **) &dev_arr_width,N*sizeof(double));
 hipMalloc((void **) &dev_arr_height,N*sizeof(double));
 hipMalloc((void **) &dev_arr_depth,N*sizeof(double));
 hipMalloc((void **) &dev_x_interval,1*4*sizeof(double));
 hipMalloc((void **) &dev_y_interval,1*4*sizeof(double));



 hipMemcpy(dev_halfvelocity,scale_halfvelocity,5*sumpart*sizeof(double),hipMemcpyHostToDevice);
 hipMemcpy(dev_velocity,scale_velocity,5*sumpart*sizeof(double),hipMemcpyHostToDevice);
 hipMemcpy(dev_force,scale_force,5*sumpart*sizeof(double),hipMemcpyHostToDevice);
 hipMemcpy(dev_position,scale_position,5*sumpart*sizeof(double),hipMemcpyHostToDevice);
 hipMemcpy(dev_mass,mass,1*sumpart*sizeof(double),hipMemcpyHostToDevice);
 hipMemcpy(dev_radius,radius,1*sumpart*sizeof(double),hipMemcpyHostToDevice);
 hipMemcpy(dev_arr_startrow,arr_startrow,N*sizeof(int),hipMemcpyHostToDevice);
 hipMemcpy(dev_arr_lastrow,arr_lastrow,N*sizeof(int),hipMemcpyHostToDevice);
 hipMemcpy(dev_arr_startx,arr_startx,N*sizeof(double),hipMemcpyHostToDevice);
 hipMemcpy(dev_arr_starty,arr_starty,N*sizeof(double),hipMemcpyHostToDevice);
 hipMemcpy(dev_arr_startz,arr_startz,N*sizeof(double),hipMemcpyHostToDevice);
 hipMemcpy(dev_arr_width,arr_width,N*sizeof(double),hipMemcpyHostToDevice);
 hipMemcpy(dev_arr_height,arr_height,N*sizeof(double),hipMemcpyHostToDevice);
 hipMemcpy(dev_arr_depth,arr_depth,N*sizeof(double),hipMemcpyHostToDevice);
 hipMemcpy(dev_x_interval,x_interval,1*4*sizeof(double),hipMemcpyHostToDevice);
 hipMemcpy(dev_y_interval,y_interval,1*4*sizeof(double),hipMemcpyHostToDevice);


 printf("\nInitial Force-------------------\n");
 for(i=0;i<5*sumpart;i+=5)
 printf("%d %d %f %f %f\n",(int)scale_force[i],(int)scale_force[i+1],scale_force[i+2],scale_force[i+3],scale_force[i+4]);


 for(t=0;t<total_time;t++)
 {
  velocityverlet<<<N,1>>>(dev_x_interval,dev_y_interval,dev_halfvelocity,dev_velocity,dev_force,dev_position,dev_mass,dev_radius,dev_arr_startrow,dev_arr_lastrow,npart,dev_arr_startx,dev_arr_starty,dev_arr_startz,dev_arr_width,dev_arr_height,dev_arr_depth);
  //gpuErrchk( hipPeekAtLastError() );
  //gpuErrchk( hipDeviceSynchronize() );
  hipDeviceSynchronize();
  hipDeviceSynchronize();

  hipMemcpy(scale_position,dev_position,sumpart*5*sizeof(double),hipMemcpyDeviceToHost);
  hipMemcpy(scale_force,dev_force,sumpart*5*sizeof(double),hipMemcpyDeviceToHost);
  hipMemcpy(scale_velocity,dev_velocity,sumpart*5*sizeof(double),hipMemcpyDeviceToHost);
  hipMemcpy(scale_halfvelocity,dev_halfvelocity,sumpart*5*sizeof(double),hipMemcpyDeviceToHost);

  printf("\n%d Force be4 exchange -------------------\n",t);
  for(i=0;i<5*sumpart;i+=5)
  printf("%d %d %f %f %f\n",(int)scale_force[i],(int)scale_force[i+1],scale_force[i+2],scale_force[i+3],scale_force[i+4]);
 
  printf("\n%d Position be4 exchange -------------------\n",t);
  for(i=0;i<5*sumpart;i+=5)
  printf("%d %d %f %f %f\n",(int)scale_position[i],(int)scale_position[i+1],scale_position[i+2],scale_position[i+3],scale_position[i+4]);

  printf("\n%d Velocity be4 exchange -------------------\n",t);
  for(i=0;i<5*sumpart;i+=5)
  printf("%d %d %f %f %f\n",(int)scale_velocity[i],(int)scale_velocity[i+1],scale_velocity[i+2],scale_velocity[i+3],scale_velocity[i+4]);

  printf("\n%d HalfVelocity be4 exchange -------------------\n",t);
  for(i=0;i<5*sumpart;i+=5)
  printf("%d %d %f %f %f\n",(int)scale_halfvelocity[i],(int)scale_halfvelocity[i+1],scale_halfvelocity[i+2],scale_halfvelocity[i+3],scale_halfvelocity[i+4]);

  printf("\n%d Arr_startrow bef4 exchange--------------------\n",t);
  for(i=0;i<N;i++)
  printf("%d ",arr_startrow[i]);
 
  printf("\n%d Arr_lastrow bef4 exchange--------------------\n",t);
  for(i=0;i<N;i++)
  printf("%d ",arr_lastrow[i]);
  
  /*for(i=0;i<npart;i++) // keep organelle posiiton constant and organelle velocity to 0
  {
   if(organelle[i]==1)
   {
    for(j=0;j<npart*5;j+=5)
    {
     if(((int)scale_position[j+1])==i)
     {
      scale_position[j+0]=organelle_pos[i][0];
      //scale_position[(j*5)+1]=organelle_pos[i][1];
      scale_position[j+2]=organelle_pos[i][2];
      scale_position[j+3]=organelle_pos[i][3];
      scale_position[j+4]=organelle_pos[i][4];
      scale_halfvelocity[j+0]=organelle_pos[i][0];
      scale_halfvelocity[j+2]=0;scale_halfvelocity[j+3]=0;scale_halfvelocity[j+4]=0;
      scale_velocity[j+0]=organelle_pos[i][0];
      scale_velocity[j+2]=0;scale_velocity[j+3]=0;scale_velocity[j+4]=0;
     }
    }
   }
  }*/

/******* rearranging scale_position according to core nos i.e., using 1st column of scale_position *********************/
  count=0;count_core=0;
  for(rearr=0;rearr<N;rearr++)
  {
   for(j=0;j<5*npart;j+=5)
   {
    if(scale_position[j]==rearr)
    {
     count_core++;
     dup_scale_position[count]=scale_position[j];
     dup_scale_force[count]=scale_force[j];
     dup_scale_velocity[count]=scale_velocity[j];
     dup_scale_halfvelocity[count]=scale_halfvelocity[j];
     count++;
     dup_scale_position[count]=scale_position[j+1];
     dup_scale_force[count]=scale_force[j+1];
     dup_scale_velocity[count]=scale_velocity[j+1];
     dup_scale_halfvelocity[count]=scale_halfvelocity[j+1];
     count++;
     dup_scale_position[count]=scale_position[j+2];
     dup_scale_force[count]=scale_force[j+2];
     dup_scale_velocity[count]=scale_velocity[j+2];
     dup_scale_halfvelocity[count]=scale_halfvelocity[j+2];
     count++;
     dup_scale_position[count]=scale_position[j+3];
     dup_scale_force[count]=scale_force[j+3];
     dup_scale_velocity[count]=scale_velocity[j+3];
     dup_scale_halfvelocity[count]=scale_halfvelocity[j+3];
     count++;
     dup_scale_position[count]=scale_position[j+4];
     dup_scale_force[count]=scale_force[j+4];
     dup_scale_velocity[count]=scale_velocity[j+4];
     dup_scale_halfvelocity[count]=scale_halfvelocity[j+4];
     count++;
    }
   }
   arr_lastrow[rearr]=count_core;
  }
  
  arr_startrow[0]=0;
  for(rearr=1;rearr<N;rearr++)
  {
   arr_startrow[rearr]=arr_lastrow[rearr-1];
  }



  fprintf(fp,"%d\n",npart);
  for(j=0;j<5*npart;j+=5)
  {
    fprintf(fp,"Atom%d ",(int)dup_scale_position[j+1]);
    fprintf(fp,"%f %f %f\n",dup_scale_position[j+2],dup_scale_position[j+3],dup_scale_position[j+4]);
   
  }

  printf("\n%d Force after exchange -------------------\n",t);
  for(i=0;i<5*sumpart;i+=5)
  printf("%d %d %f %f %f\n",(int)dup_scale_force[i],(int)dup_scale_force[i+1],dup_scale_force[i+2],dup_scale_force[i+3],dup_scale_force[i+4]);

  printf("\n%d Position after exchange -------------------\n",t);
  for(i=0;i<5*sumpart;i+=5)
  printf("%d %d %f %f %f\n",(int)dup_scale_position[i],(int)dup_scale_position[i+1],dup_scale_position[i+2],dup_scale_position[i+3],dup_scale_position[i+4]);

  printf("\n%d Arr_startrow after exchange--------------------\n",t);
  for(i=0;i<N;i++)
  printf("%d ",arr_startrow[i]);

  printf("\n%d Arr_lastrow after exchange--------------------\n",t);
  for(i=0;i<N;i++)
  printf("%d ",arr_lastrow[i]);



  hipMemcpy(dev_position,dup_scale_position,5*sumpart*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(dev_arr_startrow,arr_startrow,N*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(dev_arr_lastrow,arr_lastrow,N*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(dev_halfvelocity,dup_scale_halfvelocity,5*sumpart*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(dev_velocity,dup_scale_velocity,5*sumpart*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(dev_force,dup_scale_force,5*sumpart*sizeof(double),hipMemcpyHostToDevice);

 }

 fclose(fp);
 free(particle);free(mass);free(radius);
 free(scale_force);free(scale_halfvelocity);
 free(scale_velocity);free(scale_position);
 hipFree(dev_halfvelocity);hipFree(dev_velocity);hipFree(dev_force);hipFree(dev_position);hipFree(dev_mass);hipFree(dev_radius);
 hipFree(dev_arr_startrow);hipFree(dev_arr_lastrow);hipFree(dev_arr_startx);hipFree(dev_arr_starty);hipFree(dev_arr_startz);
 hipFree(dev_arr_width);hipFree(dev_arr_height);hipFree(dev_arr_depth);
 hipDeviceReset();
 return 0;

}

 
 
