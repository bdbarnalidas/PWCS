#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h> 
#include <time.h>
#include <string.h>
#include <math.h>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


#define G 0.01 // gravitational constant
#define delta_t 0.05 // timestep of integration
int total_time=6000; // time of integration
double rangex=0,rangey=0,rangez=0; // ranges in x and y and z direction
#define row 2 // no of rows
#define col 2 // no of columns
int boxcount=0; // count the boxes
int boxes;
double width=10; // width of each box
double height=10; // height of each box 
double depth=20; // depth of each box
double widthc=10; // width of each box copy
double heightc=10; // height of each box copy
double depthc=20; // depth of each box copy
double rad=0.5; // modify boxes according to the radius of the particle
double startx=0;
double starty=0;
double startz=0;
int npart; // no of particles
int sumpart=0; // sum of the no of particles
#define N 4 // launch cuda cores = no of boxes in the cell
int arr_startx[N],arr_starty[N],arr_startz[N];
int arr_width[N],arr_height[N],arr_depth[N]; 
int arr_startrow[N],arr_lastrow[N];

#include "cuda_wrp.c"
#include "vectdefs.h"
int x[NDIM]; // boxvector in x-direction
int y[NDIM]; // boxvector in y-direction
int z[NDIM]; // boxvector in z-direction
#include "maths.c"
#include "force.c"
#include "verlet.c"

int main()
{
 boxes=row*col; // total no of boxes
 int i=0,j=0,temporary,seed=6,k_2=0,k_3=0,k_4=0,count=0,colcount=0;
 int t=0;
 //int startrow=0,lastrow=0;
 double xr; // xr will store the random number
 FILE *fp;
 fp=fopen("output.xyz","w+"); // output coordinate file
 //int rearr=0,count_core=0;

/*************************************** Boundary boxes ***************************************************************************************/
/* 
 int x_interval[4];
 int y_interval[4];
 x_interval[0]=startx;
 y_interval[0]=starty;
 x_interval[1]=width;
 y_interval[1]=height;
 int calx;
 calx=col*width+(col-1);
 int caly;
 caly=row*height+(row-1);
 x_interval[2]=calx-width;x_interval[3]=calx;
 y_interval[2]=caly-height;y_interval[3]=caly;
 printf("x_interval\n");
 for(i=0;i<4;i++)
 printf("%d ",x_interval[i]);
 printf("\n");
 printf("y_interval\n");
 for(i=0;i<4;i++)
 printf("%d ",y_interval[i]);
 printf("\n");
*/
/*************************** Dynamically allocate particle matrix which stores no of particles of boxes ***************************************/
 int *particle = (int *)malloc(boxes*sizeof(int));
 for(i=0;i<boxes;i++)
 {
  srand(seed);
  temporary = (rand() % 10);
  if(temporary==0) // npart should not be 0 and try to keep the min no of particles to be 2
  temporary+=2;
  else if(temporary==1)
  temporary++;
  particle[i]=temporary; 
  seed++;
 }
/****************************************** Print particle matrix  ***************************************************************************/
 
 printf("Printing particle matrix --------------------------\n");
 for(i=0;i<boxes;i++)
 printf("%d ",particle[i]);
 printf("\n");

/****************************************** Allocate arr_startrow ************************************************************/
 arr_startrow[0]=0;
 for(i=1;i<N;i++)
 {
  arr_startrow[i]=arr_startrow[i-1]+particle[i-1];
 }

 printf("Printing arr_startrow--------\n");
 for(i=0;i<N;i++)
 printf("%d ",arr_startrow[i]);
 printf("\n");

/****************************************** Sum particle matrix  *****************************************************************************/
 for(i=0;i<boxes;i++)
 sumpart+=particle[i];
 //printf("%d\n",sumpart);
 npart=sumpart;
/****************************************** Allocate arr_lastrow ***************************************************************************/
 arr_lastrow[N-1]=npart-1;
 for(i=N-2;i>=0;i--)
 arr_lastrow[i]=arr_lastrow[i+1]-particle[i+1];
 for(i=0;i<N;i++)
 arr_lastrow[i]=arr_lastrow[i]+1;
 
 printf("Printing arr_lastrow--------\n");
 for(i=0;i<N;i++)
 printf("%d ",arr_lastrow[i]);
 printf("\n");

/****************************************** Dynamically allocate entire box position matrix **************************************************/
 double **position = (double **)malloc(sumpart*sizeof(double *));
 for(i=0;i<sumpart;i++)
 position[i]= (double *)malloc(5*sizeof(double));
 for(i=0;i<sumpart;i++)
 {
  for(j=0;j<5;j++)
  position[i][j]=0;
 }

/****************************************** Dynamically allocate entire force  matrix **************************************************/

 double **force = (double **)malloc(sumpart*sizeof(double *)); // dynamically allocate the force matrix
 for(i=0;i<sumpart;i++)
 force[i]= (double *)malloc(5*sizeof(double));
 for(i=0;i<sumpart;i++) // initialize the force matrix to 0
 {
  for(j=0;j<5;j++)
  {
   force[i][j]=0;
  }
 }

/****************************************** Dynamically allocate entire velocity matrix **************************************************/

 double **velocity = (double **)malloc(sumpart*sizeof(double *)); // dynamically allocate the velocity matrix
 for(i=0;i<sumpart;i++)
 velocity[i]= (double *)malloc(5*sizeof(double));
 for(i=0;i<sumpart;i++) // initialize the velocity matrix to 0
 {
  for(j=0;j<5;j++)
  {
   velocity[i][j]=0;
  }
 }

/****************************************** Dynamically allocate entire halfvelocity matrix **************************************************/

 double **halfvelocity = (double **)malloc(sumpart*sizeof(double *)); // dynamically allocate the half-velocity matrix
 for(i=0;i<sumpart;i++)
 halfvelocity[i]= (double *)malloc(5*sizeof(double));
 for(i=0;i<sumpart;i++) // initialize the half-velocity matrix to 0
 {
  for(j=0;j<5;j++)
  {
   halfvelocity[i][j]=0;
  }
 }

/****************************************** Fill the 1st column of the position matrix ****************************************************/
 boxcount=0;j=0;
 while(boxcount<boxes)
 {
  for(i=0;i<particle[boxcount];i++)
  {
   position[j][0]=boxcount;
   force[j][0]=boxcount;
   velocity[j][0]=boxcount;
   halfvelocity[j][0]=boxcount;
   j++;
  }
  boxcount++;
 }
/****************************************** Fill the 2nd column of the position matrix ****************************************************/
 j=0;
 for(i=0;i<sumpart;i++)
 {
  position[j][1]=i;
  force[j][1]=i;
  velocity[j][1]=i;
  halfvelocity[j][1]=i;
  j++;
 }
/************************************************* Test print the position matrix ************************************************************/ 
 /*for(i=0;i<sumpart;i++)
 {
  for(j=0;j<5;j++)
  printf("%f ",position[i][j]);
  printf("\n");
 } 
 printf("\n\n");*/  
/****************************************** Generate random coordinates for all the boxes ****************************************************/

 i=0;boxcount=0;k_2=0;k_3=0;k_4=0;
 while(i<row)
 {
  colcount=0;
  while(colcount<col)
  {
   x[0]=startx;x[1]=width;x[2]=0;
   y[0]=starty;y[1]=height;y[2]=0;
   z[0]=startz;z[1]=depth;z[2]=0;
   rangex=abs(x[1]-x[0]);
   rangey=abs(y[1]-y[0]);
   rangez=abs(z[1]-z[0]);
   for(j=0;j<particle[boxcount];j++) // generate x-coordinates
   {
    xr = (float)rand()/(float)(RAND_MAX/rangex)+x[0];
    position[k_2][2]=xr;k_2++;
   }
   for(j=0;j<particle[boxcount];j++) // generate y-coordinates
   {
    xr = (float)rand()/(float)(RAND_MAX/rangey)+y[0];
    position[k_3][3]=xr;k_3++;
   }
   for(j=0;j<particle[boxcount];j++) // generate z-coordinates
   {
    xr = (float)rand()/(float)(RAND_MAX/rangez)+z[0];
    position[k_4][4]=xr;k_4++;
   }
   boxcount++;
   colcount++;
   startx=width+1;width=startx+widthc;
  }
  i++;startx=0;width=widthc;starty=height+1;height=starty+heightc;  
 }
/************************************************* Test print the position matrix ************************************************************/ 
 
 for(i=0;i<sumpart;i++)
 {
  for(j=0;j<5;j++)
  printf("%f ",position[i][j]);
  printf("\n");
 }
 
/************************************************* Write the file ************************************************************/
 fprintf(fp,"%d\n",sumpart); 
 for(i=0;i<sumpart;i++)
 {
  fprintf(fp,"Atom%d ",i);
  fprintf(fp,"%f %f %f\n",position[i][2],position[i][3],position[i][4]);
 }  

/*************************** Dynamically allocate organelle_pos matrix which stores coordinates of organelles ***********************************/
 double **organelle_pos = (double **)malloc(npart*sizeof(double *));
 for(i=0;i<npart;i++)
 organelle_pos[i]= (double *)malloc(NDIM*sizeof(double));

 for(i=0;i<npart;i++)
 {
  for(j=0;j<NDIM;j++)
  organelle_pos[i][j]=0;
 }

/*************************Dynamically allocate mass matrix which stores masses of particles ***************************************/
 double *mass = (double *)malloc(sumpart*sizeof(double));
 for(i=0;i<sumpart;i++)
 mass[i]=1; // all the particles are of mass 1

 mass[1]=999; // organelle
 mass[7]=999;
 mass[12]=999;
 mass[15]=999;

/*************************** Dynamically allocate radius matrix which stores radii of particles **************************************/
 double *radius = (double *)malloc(sumpart*sizeof(double));
 for(i=0;i<sumpart;i++)
 radius[i]=0.5; // all the particles are of radius 0.5

 radius[1]=1.0; // organelle
 radius[7]=1.0;
 radius[12]=1.0;
 radius[15]=1.0;

/*************************** Dynamically allocate organelle matrix which stores the organelle info **************************************/
 double *organelle = (double *)malloc(npart*sizeof(double));
 for(i=0;i<npart;i++)
 organelle[i]=0; // initialise the organelle array to 0 i.e., no organelles are present

 organelle[1]=1; // this particle is an organelle
 organelle[7]=1;
 organelle[12]=1;
 organelle[15]=1;
 //organelle[5]=1;
 //organelle[6]=1;
 //organelle[11]=1;

/******************************** store coordinates of organelles into organelle_pos matrix ***********************************/
 for(i=0;i<npart;i++)
 {
  if(organelle[i]==1)
  {
   organelle_pos[i][0]=position[i][2];
   organelle_pos[i][1]=position[i][3];
   organelle_pos[i][2]=position[i][4];
  }
 }

/*************************** Generate the force matrix *******************************************************************************/
 
 force=vanderwalforce(force,position,mass,npart); // to calculate vanderwal force
 printf("\n\n FORCE MATRIX ----------------\n");
 for(i=0;i<sumpart;i++)
 {
  for(j=0;j<5;j++)
  printf("%f ",force[i][j]);
  printf("\n");
 }

/************************* Allocate arr_startx,arr_starty,arr_startz,arr_width,arr_height,arr_depth *****************************************************/
 i=0;boxcount=0;startx=0;width=widthc;starty=0;height=heightc;startz=0;depth=depthc;
 while(i<row)
 {
  colcount=0;
  while(colcount<col)
  {
   arr_startx[boxcount]=startx;
   arr_starty[boxcount]=starty;
   arr_startz[boxcount]=startz;
   arr_width[boxcount]=width;
   arr_height[boxcount]=height;
   arr_depth[boxcount]=depth;
   boxcount++;
   colcount++;
   startx=width+1;width=startx+widthc;
  }
  i++;startx=0;width=widthc;starty=height+1;height=starty+heightc; 
 }

 printf("Printing arr_startx-------------------------\n");
 for(i=0;i<N;i++)
 printf("%d ",arr_startx[i]);
 printf("\n");
 printf("Printing arr_starty-------------------------\n");
 for(i=0;i<N;i++)
 printf("%d ",arr_starty[i]);
 printf("\n");
 printf("Printing arr_startz-------------------------\n");
 for(i=0;i<N;i++)
 printf("%d ",arr_startz[i]);
 printf("\n");
 printf("Printing arr_width-------------------------\n");
 for(i=0;i<N;i++)
 printf("%d ",arr_width[i]);
 printf("\n");
 printf("Printing arr_height-------------------------\n");
 for(i=0;i<N;i++)
 printf("%d ",arr_height[i]);
 printf("\n");
 printf("Printing arr_depth-------------------------\n");
 for(i=0;i<N;i++)
 printf("%d ",arr_depth[i]);
 printf("\n");













/************************* Convert main 2D arrays to 1D arrays to ease cuda processing ******************************************************************/
 
 double *scale_halfvelocity;// 1D scaled version of halfvelocity        host copies
 double *scale_velocity;// 1D scaled version of velocity
 double *scale_force;// 1D scaled version of force
 double *scale_position;// 1D scaled version of position
 double *dup_scale_position,*dup_scale_halfvelocity,*dup_scale_velocity,*dup_scale_force;

 scale_halfvelocity=(double *)malloc(5*sumpart*sizeof(double));
 scale_velocity=(double *)malloc(5*sumpart*sizeof(double));
 scale_force=(double *)malloc(5*sumpart*sizeof(double));
 scale_position=(double *)malloc(5*sumpart*sizeof(double));
 dup_scale_position=(double *)malloc(5*sumpart*sizeof(double));
 dup_scale_halfvelocity=(double *)malloc(5*sumpart*sizeof(double));
 dup_scale_velocity=(double *)malloc(5*sumpart*sizeof(double));
 dup_scale_force=(double *)malloc(5*sumpart*sizeof(double));



 
 count=0;
 for(i=0;i<sumpart;i++) // Allocating halfvelocity matrix to scale_halfvelocity array
 {
  for(j=0;j<5;j++)
  {
   scale_halfvelocity[count]=halfvelocity[i][j];
   count++;
  }
 }


 
 count=0;
 for(i=0;i<sumpart;i++) // Allocating velocity matrix to scale_velocity array
 {
  for(j=0;j<5;j++)
  {
   scale_velocity[count]=velocity[i][j];
   count++;
  }
 }

 count=0;
 for(i=0;i<sumpart;i++) // Allocating force matrix to scale_force array
 {
  for(j=0;j<5;j++)
  {
   scale_force[count]=force[i][j];
   count++;
  }
 }

 count=0;
 for(i=0;i<sumpart;i++) // Allocating position matrix to scale_position array
 {
  for(j=0;j<5;j++)
  {
   scale_position[count]=position[i][j];
   count++;
  }
 }

 count=0;
 for(i=0;i<sumpart;i++) // Initialising dup_scale_position 
 {
  for(j=0;j<5;j++)
  {
   dup_scale_position[count]=0;
   count++;
  }
 }
 
 count=0;
 for(i=0;i<sumpart;i++) // Initialising dup_scale_halfvelocity 
 {
  for(j=0;j<5;j++)
  {
   dup_scale_halfvelocity[count]=0;
   count++;
  }
 }


 count=0;
 for(i=0;i<sumpart;i++) // Initialising dup_scale_velocity 
 {
  for(j=0;j<5;j++)
  {
   dup_scale_velocity[count]=0;
   count++;
  }
 }

 count=0;
 for(i=0;i<sumpart;i++) // Initialising dup_scale_force 
 {
  for(j=0;j<5;j++)
  {
   dup_scale_force[count]=0;
   count++;
  }
 }



 free(halfvelocity);free(velocity);free(force);free(position);

 int pupu;
/************************* Device initialisations and allocating space **********************************************************************************/
 //hipDeviceReset();
 double *dev_halfvelocity,*dev_velocity,*dev_force,*dev_position,*dev_mass,*dev_radius,*dev_organelle;// device copies
 int *dev_arr_startrow,*dev_arr_lastrow,*dev_arr_startx,*dev_arr_starty,*dev_arr_startz,*dev_arr_width,*dev_arr_height,*dev_arr_depth;
 int *dev_x_interval,*dev_y_interval;
 int dev_pupu;
 hipMalloc((void *) &dev_pupu,sizeof(int));

 hipMalloc((void **) &dev_halfvelocity,5*sumpart*sizeof(double));
 hipMalloc((void **) &dev_velocity,5*sumpart*sizeof(double));
 hipMalloc((void **) &dev_force,5*sumpart*sizeof(double));
 hipMalloc((void **) &dev_position,5*sumpart*sizeof(double));
 hipMalloc((void **) &dev_mass,1*sumpart*sizeof(double));
 hipMalloc((void **) &dev_radius,1*sumpart*sizeof(double));
 hipMalloc((void **) &dev_arr_startrow,N*sizeof(int));
 hipMalloc((void **) &dev_arr_lastrow,N*sizeof(int));
 hipMalloc((void **) &dev_arr_startx,N*sizeof(int));
 hipMalloc((void **) &dev_arr_starty,N*sizeof(int));
 hipMalloc((void **) &dev_arr_startz,N*sizeof(int));
 hipMalloc((void **) &dev_arr_width,N*sizeof(int));
 hipMalloc((void **) &dev_arr_height,N*sizeof(int));
 hipMalloc((void **) &dev_arr_depth,N*sizeof(int));
 hipMalloc((void **) &dev_x_interval,1*4*sizeof(int));
 hipMalloc((void **) &dev_y_interval,1*4*sizeof(int));
 hipMalloc((void **) &dev_organelle,1*sumpart*sizeof(double));



 hipMemcpy(dev_halfvelocity,scale_halfvelocity,5*sumpart*sizeof(double),hipMemcpyHostToDevice);
 hipMemcpy(dev_velocity,scale_velocity,5*sumpart*sizeof(double),hipMemcpyHostToDevice);
 hipMemcpy(dev_force,scale_force,5*sumpart*sizeof(double),hipMemcpyHostToDevice);
 hipMemcpy(dev_position,scale_position,5*sumpart*sizeof(double),hipMemcpyHostToDevice);
 hipMemcpy(dev_mass,mass,1*sumpart*sizeof(double),hipMemcpyHostToDevice);
 hipMemcpy(dev_radius,radius,1*sumpart*sizeof(double),hipMemcpyHostToDevice);
 hipMemcpy(dev_arr_startrow,arr_startrow,N*sizeof(int),hipMemcpyHostToDevice);
 hipMemcpy(dev_arr_lastrow,arr_lastrow,N*sizeof(int),hipMemcpyHostToDevice);
 hipMemcpy(dev_arr_startx,arr_startx,N*sizeof(int),hipMemcpyHostToDevice);
 hipMemcpy(dev_arr_starty,arr_starty,N*sizeof(int),hipMemcpyHostToDevice);
 hipMemcpy(dev_arr_startz,arr_startz,N*sizeof(int),hipMemcpyHostToDevice);
 hipMemcpy(dev_arr_width,arr_width,N*sizeof(int),hipMemcpyHostToDevice);
 hipMemcpy(dev_arr_height,arr_height,N*sizeof(int),hipMemcpyHostToDevice);
 hipMemcpy(dev_arr_depth,arr_depth,N*sizeof(int),hipMemcpyHostToDevice);
 //hipMemcpy(dev_x_interval,x_interval,1*4*sizeof(int),hipMemcpyHostToDevice);
 //hipMemcpy(dev_y_interval,y_interval,1*4*sizeof(int),hipMemcpyHostToDevice);
 hipMemcpy(dev_organelle,organelle,1*sumpart*sizeof(double),hipMemcpyHostToDevice);


 printf("\nInitial Force-------------------\n");
 for(i=0;i<5*sumpart;i+=5)
 printf("%d %d %f %f %f\n",(int)scale_force[i],(int)scale_force[i+1],scale_force[i+2],scale_force[i+3],scale_force[i+4]);


 for(t=0;t<total_time;t++)
 {
  velocityverlet<<<N,1>>>(dev_halfvelocity,dev_velocity,dev_force,dev_position,dev_mass,dev_radius,dev_arr_startrow,dev_arr_lastrow,npart,dev_arr_startx,dev_arr_starty,dev_arr_startz,dev_arr_width,dev_arr_height,dev_arr_depth,dev_organelle,dev_pupu);
  //gpuErrchk( hipPeekAtLastError() );
  //gpuErrchk( hipDeviceSynchronize() );
  hipDeviceSynchronize();
  hipDeviceSynchronize();

  hipMemcpy(scale_position,dev_position,sumpart*5*sizeof(double),hipMemcpyDeviceToHost);
  hipMemcpy(scale_force,dev_force,sumpart*5*sizeof(double),hipMemcpyDeviceToHost);
  hipMemcpy(scale_velocity,dev_velocity,sumpart*5*sizeof(double),hipMemcpyDeviceToHost);
  hipMemcpy(scale_halfvelocity,dev_halfvelocity,sumpart*5*sizeof(double),hipMemcpyDeviceToHost);
  hipMemcpy(pupu,dev_pupu,sizeof(int),hipMemcpyDeviceToHost);

  printf("\n%d Force be4 exchange -------------------\n",t);
  for(i=0;i<5*sumpart;i+=5)
  printf("%d %d %f %f %f\n",(int)scale_force[i],(int)scale_force[i+1],scale_force[i+2],scale_force[i+3],scale_force[i+4]);

  printf("\n%d Position be4 exchange -------------------\n",t);
  for(i=0;i<5*sumpart;i+=5)
  printf("%d %d %f %f %f\n",(int)scale_position[i],(int)scale_position[i+1],scale_position[i+2],scale_position[i+3],scale_position[i+4]);

  printf("\n%d Velocity be4 exchange -------------------\n",t);
  for(i=0;i<5*sumpart;i+=5)
  printf("%d %d %f %f %f\n",(int)scale_velocity[i],(int)scale_velocity[i+1],scale_velocity[i+2],scale_velocity[i+3],scale_velocity[i+4]);

  printf("\n%d HalfVelocity be4 exchange -------------------\n",t);
  for(i=0;i<5*sumpart;i+=5)
  printf("%d %d %f %f %f\n",(int)scale_halfvelocity[i],(int)scale_halfvelocity[i+1],scale_halfvelocity[i+2],scale_halfvelocity[i+3],scale_halfvelocity[i+4]);


  
  for(i=0;i<npart;i++) // keep organelle position constant
  {
   if(organelle[i]==1)
   {
    scale_position[(i*5)+2]=organelle_pos[i][0];
    scale_position[(i*5)+3]=organelle_pos[i][1];
    scale_position[(i*5)+4]=organelle_pos[i][2];
   }
  }

  for(i=0;i<npart;i++) // keep organelle velocity 0
  {
   if(organelle[i]==1)
   {
    scale_halfvelocity[(i*5)+2]=0;
    scale_halfvelocity[(i*5)+3]=0;
    scale_halfvelocity[(i*5)+4]=0;
    scale_velocity[(i*5)+2]=0;  
    scale_velocity[(i*5)+3]=0;
    scale_velocity[(i*5)+4]=0;
   }
  }  


  fprintf(fp,"%d\n",npart);count=0; // print position to file
  for(j=0;j<5*npart;j+=5)
  {
   fprintf(fp,"Atom%d ",count);
   fprintf(fp,"%f %f %f\n",scale_position[j+2],scale_position[j+3],scale_position[j+4]);
   count++;
  }
  hipMemcpy(dev_halfvelocity,scale_halfvelocity,5*sumpart*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(dev_velocity,scale_velocity,5*sumpart*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(dev_position,scale_position,5*sumpart*sizeof(double),hipMemcpyHostToDevice);

 
  /*int pupu;
  for(j=0;j<5*npart;j+=5)
  {
   pupu=(int)scale_position[j+1];
   if(organelle[pupu]==1)
   {
      scale_position[j+0]=organelle_pos[pupu][0];
      //scale_position[(j*5)+1]=organelle_pos[i][1];
      scale_position[j+2]=organelle_pos[pupu][2];
      scale_position[j+3]=organelle_pos[pupu][3];
      scale_position[j+4]=organelle_pos[pupu][4];
      scale_halfvelocity[j+0]=organelle_pos[pupu][0];
      scale_halfvelocity[j+2]=0;scale_halfvelocity[j+3]=0;scale_halfvelocity[j+4]=0;
      scale_velocity[j+0]=organelle_pos[pupu][0];
      scale_velocity[j+2]=0;scale_velocity[j+3]=0;scale_velocity[j+4]=0;
      scale_force[j+0]=organelle_pos[pupu][0];
   }
  }*/

/******* rearranging scale_position according to core nos i.e., using 1st column of scale_position *********************/
  /*count=0;count_core=0;
  for(rearr=0;rearr<N;rearr++)
  {
   for(j=0;j<5*npart;j+=5)
   {
    if(scale_position[j]==rearr)
    {
     count_core++;
     dup_scale_position[count]=scale_position[j];
     dup_scale_force[count]=scale_force[j];
     dup_scale_velocity[count]=scale_velocity[j];
     dup_scale_halfvelocity[count]=scale_halfvelocity[j];
     count++;
     dup_scale_position[count]=scale_position[j+1];
     dup_scale_force[count]=scale_force[j+1];
     dup_scale_velocity[count]=scale_velocity[j+1];
     dup_scale_halfvelocity[count]=scale_halfvelocity[j+1];
     count++;
     dup_scale_position[count]=scale_position[j+2];
     dup_scale_force[count]=scale_force[j+2];
     dup_scale_velocity[count]=scale_velocity[j+2];
     dup_scale_halfvelocity[count]=scale_halfvelocity[j+2];
     count++;
     dup_scale_position[count]=scale_position[j+3];
     dup_scale_force[count]=scale_force[j+3];
     dup_scale_velocity[count]=scale_velocity[j+3];
     dup_scale_halfvelocity[count]=scale_halfvelocity[j+3];
     count++;
     dup_scale_position[count]=scale_position[j+4];
     dup_scale_force[count]=scale_force[j+4];
     dup_scale_velocity[count]=scale_velocity[j+4];
     dup_scale_halfvelocity[count]=scale_halfvelocity[j+4];
     count++;
    }
   }
   arr_lastrow[rearr]=count_core;
  }
  
  arr_startrow[0]=0;
  for(rearr=1;rearr<N;rearr++)
  {
   arr_startrow[rearr]=arr_lastrow[rearr-1];
  }

  int barnali;
  fprintf(fp,"%d\n",npart);
  for(j=0;j<5*npart;j+=5)
  {
   barnali=(int)dup_scale_position[j+1];
   if(organelle[barnali]==1)
   {
    fprintf(fp,"H ");
    //fprintf(fp,"Atom%d ",(int)dup_scale_position[j+1]);
    fprintf(fp,"%f %f %f\n",dup_scale_position[j+2],dup_scale_position[j+3],dup_scale_position[j+4]);
   }
   else
   {
    fprintf(fp,"Atom%d ",(int)dup_scale_position[j+1]);
    fprintf(fp,"%f %f %f\n",dup_scale_position[j+2],dup_scale_position[j+3],dup_scale_position[j+4]);
   }
  }*/



  
  /*hipMemcpy(dev_position,dup_scale_position,5*sumpart*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(dev_arr_startrow,arr_startrow,N*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(dev_arr_lastrow,arr_lastrow,N*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(dev_halfvelocity,dup_scale_halfvelocity,5*sumpart*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(dev_velocity,dup_scale_velocity,5*sumpart*sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(dev_force,dup_scale_force,5*sumpart*sizeof(double),hipMemcpyHostToDevice);*/


 }

 fclose(fp);
 free(particle);free(mass);free(radius);
 free(scale_force);free(scale_halfvelocity);
 free(scale_velocity);free(scale_position);
 hipFree(dev_halfvelocity);hipFree(dev_velocity);hipFree(dev_force);hipFree(dev_position);hipFree(dev_mass);hipFree(dev_radius);
 hipFree(dev_arr_startrow);hipFree(dev_arr_lastrow);hipFree(dev_arr_startx);hipFree(dev_arr_starty);hipFree(dev_arr_startz);
 hipFree(dev_arr_width);hipFree(dev_arr_height);hipFree(dev_arr_depth);
 hipDeviceReset();
 return 0;

}

 
 
